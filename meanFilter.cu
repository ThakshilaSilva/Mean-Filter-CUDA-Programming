#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <tuple>
#include <iostream>
#include <string.h>

double time_h = 0;
double time_d = 0;

int numOfRounds = 5;

void meanFilter_h (unsigned char* raw_image, unsigned char* filtered_image, int img_width, int img_height, int window_size)
{
	int half_window = (window_size - 1) / 2;
	
	for (int i=0; i < img_height; i++)
	{
		for(int j=0; j < img_width; j++)
		{
			int left_limit, right_limit, top_limit, bottom_limit;
			
			if(j - half_window >= 0){
				left_limit = j-half_window;
			}else{
				left_limit = 0;
			}
			
            if(j + half_window <= img_width-1){
				right_limit = j + half_window;
			}else{
				right_limit = img_width-1;
			}
			
			if(i - half_window >= 0){
				top_limit = i - half_window;
			}else{
				top_limit = 0;
			}
			
            if(i + half_window <= img_height-1){
				bottom_limit = i + half_window;
			}else{
				bottom_limit = img_height-1;
			}
			
			double sum = 0;
			for(int k = top_limit; k <= bottom_limit; k++)
			{
				for(int m = left_limit; m <= right_limit; m++)
				{
					sum += raw_image[(k * img_height) + m];
				}
			}
			int current_window_size = (bottom_limit - top_limit + 1) * (right_limit - left_limit + 1);
			filtered_image[i*img_height + j] = sum / current_window_size; 
		}
	}
}

__global__ void meanFilter_d (unsigned char* raw_image, unsigned char* filtered_image, int img_width, int img_height, int window_size)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	int half_window = (window_size - 1) / 2;
	
	if (i < img_height && j < img_width)
	{
		int left_limit, right_limit, top_limit, bottom_limit;
			
		if(j - half_window >= 0){
			left_limit = j-half_window;
		}else{
			left_limit = 0;
		}
			
        if(j + half_window <= img_width-1){
			right_limit = j + half_window;
		}else{
			right_limit = img_width-1;
		}
			
		if(i - half_window >= 0){
			top_limit = i - half_window;
		}else{
			top_limit = 0;
		}
			
        if(i + half_window <= img_height-1){
			bottom_limit = i + half_window;
		}else{
			bottom_limit = img_height-1;
		}
		
		double sum = 0;
		for(int k = top_limit; k <= bottom_limit; k++)
		{
			for(int m = left_limit; m <= right_limit; m++)
			{
				sum += raw_image[(k * img_height) + m];
			}
		}
		int current_window_size = (bottom_limit - top_limit + 1) * (right_limit - left_limit + 1);
		filtered_image[i*img_height + j] = sum / current_window_size;
	}
}

int main(int argc,char **argv)
{
    printf("Begin......\n");
    
	//get bitmap to a char array
    FILE* file = fopen(argv[1], "rb");
    unsigned char info[54];
    fread(info, sizeof(unsigned char), 54, file);

    int width, height;
    memcpy(&width, info + 18, sizeof(int));
    memcpy(&height, info + 22, sizeof(int));

    int window_size = strtol(argv[2],NULL,10);
        
    int size = 3 * width * abs(height);
    unsigned char* inputImage = new unsigned char[size];
    unsigned char* result_image_data_d;
    unsigned char* result_image_data_h = new unsigned char[size];
    unsigned char* result_image_data_h1 = new unsigned char[size];

    unsigned char* image_data_d;

    fread(inputImage, sizeof(unsigned char), size, file);
    fclose(file);
 
    int block_size = 32;
    int grid_size = width/block_size;
	
    dim3 dimBlock(block_size, block_size, 1);
    dim3 dimGrid(grid_size, grid_size, 1);
    
    for(int x = 0; x < numOfRounds; x += 1)
    {
        hipMalloc((void **)&image_data_d,size*sizeof(unsigned char));
        hipMalloc((void **)&result_image_data_d,size*sizeof(unsigned char));

        hipMemcpy(image_data_d,inputImage,size*sizeof(unsigned char),hipMemcpyHostToDevice);

        clock_t start_d=clock();
		//execution of GPU code
        meanFilter_d <<< dimGrid, dimBlock >>> (image_data_d, result_image_data_d, width, height, window_size);
        hipDeviceSynchronize();

        hipError_t error = hipGetLastError();
        if(error!=hipSuccess)
        {
            fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
            exit(-1);
        }
		
        clock_t end_d = clock();

        clock_t start_h = clock();
		//executing CPU code
        meanFilter_h(inputImage, result_image_data_h1, width, height, window_size);
        clock_t end_h = clock();

        time_h += (double)(end_h-start_h)/CLOCKS_PER_SEC;
        time_d += (double)(end_d-start_d)/CLOCKS_PER_SEC;

        hipFree(image_data_d);
        hipFree(result_image_data_d);
    }

    printf("Average GPU execution time: %f\n",(time_d/numOfRounds));
    printf("Average CPU execution time: %f\n",(time_h/numOfRounds));
    printf("CPU/GPU time: %f\n",(time_h/time_d));

    return 0;
}



































